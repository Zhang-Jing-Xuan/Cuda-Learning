
#include <hip/hip_runtime.h>
#include <stdio.h>

static void HandleError(hipError_t err,const char * file,int line){
    if(err!=hipSuccess){
        printf("%s in %s at line %d\n",hipGetErrorString(err),file,line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err,__FILE__,__LINE__))

int getThreadNum(){
    hipDeviceProp_t prop;
    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    printf("gpu num %d\n",count);
    HANDLE_ERROR(hipGetDeviceProperties(&prop,0));
    printf("max thread num:%d\n",prop.maxThreadsPerBlock);
    printf("max grid dimensions:%d %d %d\n",prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

__global__ void conv(float *img,float *kernel,float *result,int width,int height,int kernelSize){
    int ti=threadIdx.x;
    int bi=blockIdx.x;
    
}
int main(){
    int width=10;
    int height=10;
    float *img=new float[width*height];
    for(int i=0;i<height;i++){
        for(int j=0;j<width;j++){
            img[j+i*width]=(i+j)%256;
        }
    }

    int kernelSize=3;
    float *kernel=new float[kernelSize*kernelSize];
    for(int i=0;i<kernelSize*kernelSize;i++){
        kernel[i]=i%kernelSize-1;
    }

    float *imgGpu,*kernelGpu,*resultGpu;

    HANDLE_ERROR(hipMalloc((void**)&imgGpu,width*height*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&kernelGpu,kernelSize*kernelSize*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&resultGpu,width*height*sizeof(float)));

    HANDLE_ERROR(hipMemcpy(imgGpu,img,width*height*sizeof(float),hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(kernelGpu,kernel,kernelSize*kernelSize*sizeof(float),hipMemcpyHostToDevice));

    int threadNum=getThreadNum();
    int blockNum=(width*height-0.5)/threadNum+1;
    conv<<<blockNum,threadNum>>>(imgGpu,kernelGpu,resultGpu,width,height,kernelSize);
    //Visualization
    printf("img:\n");
    for(int i=0;i<10;i++){
        for(int j=0;j<10;j++){
            printf("%2.0f ",img[j+i*width]);
        }
        puts("");
    }
    printf("kernel:\n");
    for(int i=0;i<kernelSize;i++){
        for(int j=0;j<kernelSize;j++){
            printf("%2.0lf ",kernel[i*kernelSize+j]);
        }
        puts("");
    }
    return 0;
}